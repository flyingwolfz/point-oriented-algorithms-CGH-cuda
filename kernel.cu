#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

__global__ void test(float *dimage, float *dholo )
{
	int h = blockIdx.x*blockDim.x + threadIdx.x;
	int l = blockIdx.y*blockDim.y + threadIdx.y;
	float x = (float)h;
	float y = (float)l;
	float hx = (x - 512.0)*0.008;
	float hy = (y - 512.0)*0.008;
	float lambda = 639.0 * 0.000001;
	float d = 0.008*10.0, holo = 0.0,  c = 0.0, s = 0.0;
	float ii = 0.0, jj = 0.0;
	long int prt = h + l * 1024;
	register float xx;
	register float yy,image,r,phi;
	register int k=0;
	float dep = 0.0f;
	 for (register int i = 0; i < 100; i++)
	 {
		 for (register int j = 0; j < 100; j++)
		 {
			 xx = ((float)i - 50.0)*d;
			 yy = ((float)j - 50.0)*d;	
			 image = dimage[k];
			 dep = 300;
			 k++;
			 r = (hx - xx)*(hx - xx) + (hy - yy)*(hy - yy) + dep*dep;
			 phi = 2 * 3.14159 / lambda * sqrt(r);
			 c = c+ image * cos(phi);
			 s = s+ image * sin(phi);	

		 }	 
	 }
	float jiao = atan2(s, c);
	if (jiao < 0)
	{
		jiao = jiao + 2.0 * 3.14159;
	}
	dholo[prt] = jiao / 2.0 / 3.14159;
}

extern "C"  void kernel(float *dimage, float *dholo)
{
	
	dim3 block(32, 32);
	dim3 grid(32, 32);

	test <<<grid, block >>> (dimage, dholo);

}